#include "hip/hip_runtime.h"
/** 
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

#include <xmmintrin.h>

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// includes, CUDA
#include <builtin_types.h>

#include <hip/hip_fp16.h>


// Complex data type
typedef half2 halfComplex;
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMulAndScale(Complex *, const Complex *,
                                                   int, float);

// Filtering functions
void Convolve(const Complex *, int, const Complex *, int, Complex *);

// Padding functions
int PadData(const Complex *, Complex **, int, const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

// The filter size is assumed to be a number smaller than the signal size
//#define SIGNAL_SIZE 50
//#define SIGNAL_SIZE 1024
//#define SIGNAL_SIZE 8192
//#define SIGNAL_SIZE 256
#define SIGNAL_SIZE 0x400000
#define BATCH       16

halfComplex h_signal_half[SIGNAL_SIZE * BATCH];
halfComplex h_convolved_signal_half[SIGNAL_SIZE * BATCH];
//Complex h_signal[SIGNAL_SIZE * BATCH];
//Complex h_convolved_signal[SIGNAL_SIZE * BATCH];

int16_t h_signal16[SIGNAL_SIZE*2*BATCH];
int16_t h_convolved_signal16[SIGNAL_SIZE*2 * BATCH];

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
    LARGE_INTEGER li;
    if (!QueryPerformanceFrequency(&li))
        printf("QueryPerformanceFrequency failed!\n");

    PCFreq = (double)(li.QuadPart) / 1000.0;

    QueryPerformanceCounter(&li);
    CounterStart = li.QuadPart;
}
double GetCounter()
{
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    return (double)(li.QuadPart - CounterStart) / PCFreq;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { runTest(argc, argv); }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  printf("[simpleCUFFT] is starting...\n");

  findCudaDevice(argc, (const char **)argv);

  // Allocate host memory for the signal
//  Complex *h_signal =
//      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));

  int mem_size = sizeof(hipfftComplex) * SIGNAL_SIZE * BATCH;
  int mem_size_half = sizeof(hipfftComplex) * (SIGNAL_SIZE/2) * BATCH;

    // host arrays
  Complex *h_PinnedSignal, *h_PinnedConvolvedSignal;
  halfComplex* h_PinnedSignal_half, * h_PinnedConvolvedSignal_half;
  // allocate and initialize
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedSignal, mem_size)); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedConvolvedSignal, mem_size)); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedSignal_half, mem_size_half)); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedConvolvedSignal_half, mem_size_half)); // host pinned


  // Initialize the memory for the signal
  halfComplex* p_half;

  p_half = (halfComplex*)h_signal16;

  for (unsigned int i = 0; i < SIGNAL_SIZE*2*BATCH; i+=2) {
      h_signal16[i] = 0;
      h_signal16[i+1] = 0;
  }

  for (unsigned int i = 0; i < BATCH * 2; i += 2) {
      h_signal16[i] = i + 1;
      h_signal16[i + 1] = 0;
  }
  /*
  for (unsigned int i = 0; i < SIGNAL_SIZE*BATCH; i++) {
      h_signal[i].x = p_half[i].x;
      h_signal[i].y = p_half[i].y;
  }
  */

  StartCounter();
  // Initialize the memory for the signal
#pragma loop(hint_parallel(0))
  for (unsigned int i = 0; i < SIGNAL_SIZE*BATCH; ++i) {
    //h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
    //h_signal[i].y = 0;
//    h_signal_half[i].x = (half)i;
  //  h_signal_half[i].y = (half)0;
    //h_signal[i].x = h_signal_half[i].x;
    //h_signal[i].y = h_signal_half[i].y;
    h_PinnedSignal_half[i].x = h_signal16[2*i];
    h_PinnedSignal_half[i].y = h_signal16[2*i+1];
  }
  
  float xx = h_PinnedSignal_half[0].x;
  xx/=32768;
  h_PinnedSignal_half[0].x = xx;
  float yy = h_PinnedSignal_half[0].x;

  double copytime = GetCounter();
  printf("----- Int2float time %ld Bytes is: %0.3f milliseconds \n", mem_size, copytime);
  printf("--------------------------------------------- \n");
  
  
  // Allocate device memory for signal
  halfComplex *d_signal;
  halfComplex* r_signal;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size_half));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&r_signal), mem_size_half));
  
  StartCounter();
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_signal, h_signal_half, mem_size_half, hipMemcpyHostToDevice));
  copytime = GetCounter();
  printf("---- Copy time %ld Bytes is: %0.3f milliseconds \n", mem_size_half, copytime);
  printf("--------------------------------------------- \n");

  StartCounter();
  // Copy host memory to device
  //checkCudaErrors(hipMemcpy(d_signal, h_PinnedSignal_half, mem_size_half, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_signal, p_half, mem_size_half, hipMemcpyHostToDevice));
  copytime = GetCounter();
  printf("---- Pinned Memory Copy time %ld Bytes is: %0.3f milliseconds \n", mem_size, copytime);
  printf("--------------------------------------------- \n");

    // CUFFT plan simple API
  hipfftHandle plan;
  //checkCudaErrors(hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 1));
  checkCudaErrors(hipfftCreate(&plan));
  int rank = 1;
  //long long int n = 65536*8*2*4;// 0x1000000;// SIGNAL_SIZE;
 /*
  long long int n = SIGNAL_SIZE/2;
  long long int inembed[] = { 0 };
  long long int istride = 1; 
  long long int idist = n;
  hipDataType inputtype = HIP_C_16F;
  long long int onembed[] = { 0 };
  long long int ostride = 1;
  long long int odist = n;
  hipDataType outputtype = HIP_C_16F;
*/
  long long int n = SIGNAL_SIZE / 2;
  long long int inembed[] = { 0 };
  long long int istride = 2;
  long long int idist = 1;
  hipDataType inputtype = HIP_C_16F;
  long long int onembed[] = { 0 };
  long long int ostride = 1;
  long long int odist = n;
  hipDataType outputtype = HIP_C_16F;

  long long int batch = BATCH;// 1;
  size_t workSize;
  hipDataType executiontype = HIP_C_16F;
  checkCudaErrors(hipfftXtMakePlanMany(plan, rank, &n,
      inembed, istride, idist, inputtype,
      onembed, ostride, odist, outputtype,
      batch, &workSize,
      executiontype));

  // Transform signal and kernel
  printf("Transforming signal hipfftXtExec\n");
  
  // timer init
  hipEvent_t start, stop;
  float gpuTime = 0.0f;
  hipEventCreate(&start, 0);
  hipEventCreate(&stop, 0);
  hipEventRecord(start, 0);
  hipEventSynchronize(start);

  StartCounter();

  checkCudaErrors(hipfftXtExec(plan, d_signal, r_signal, HIPFFT_FORWARD));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  double ttime = GetCounter();

  hipEventElapsedTime(&gpuTime, start, stop);
  printf("---- time: %.10f milliseconds\n", gpuTime);
  printf("---- Execution time is: %0.3f milliseconds \n", ttime);
  printf("--------------------------------------------- \n");

  // Copy device memory to host
  checkCudaErrors(hipMemcpy(h_convolved_signal_half, r_signal, mem_size_half,
        hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_signal_half, d_signal, mem_size_half,
      hipMemcpyDeviceToHost));

  for (int ii = 0; ii < SIGNAL_SIZE; ii++) {
      h_convolved_signal[ii].x = h_convolved_signal_half[ii].x;
      h_convolved_signal[ii].y = h_convolved_signal_half[ii].y;
  }

  // Check if kernel execution generated and error
   getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

  // Transform signal back
  printf("Transforming signal back hipfftExecC2C\n");
  checkCudaErrors(hipfftXtExec(plan, r_signal, d_signal, HIPFFT_BACKWARD));

  // Copy device memory to host
  checkCudaErrors(hipMemcpy(h_convolved_signal_half, d_signal, mem_size_half,
                             hipMemcpyDeviceToHost));

  
  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));

  // cleanup memory
  checkCudaErrors(hipFree(d_signal));
  checkCudaErrors(hipFree(r_signal));

  exit(EXIT_SUCCESS);
}


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////
/*
// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b,
                                                   int size, float scale) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = threadID; i < size; i += numThreads) {
    a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
  }
}
*/