#include "hip/hip_runtime.h"
/** 
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

#include <xmmintrin.h>

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// includes, CUDA
#include <builtin_types.h>

// Complex data type
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMulAndScale(Complex *, const Complex *,
                                                   int, float);
// Int16 to Float32
__global__ void Int16toFloat32(int16_t* a, Complex* b, int batch, long long size);

// Filtering functions
void Convolve(const Complex *, int, const Complex *, int, Complex *);

// Padding functions
int PadData(const Complex *, Complex **, int, const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

// The filter size is assumed to be a number smaller than the signal size
//#define SIGNAL_SIZE 50
//#define SIGNAL_SIZE 1024
//#define SIGNAL_SIZE 8192
//#define SIGNAL_SIZE 256
#define SIGNAL_SIZE 0x400000
#define BATCH       16

Complex h_signal[SIGNAL_SIZE * BATCH];
Complex h_convolved_signal[SIGNAL_SIZE * BATCH];

//int16_t h_signal16[SIGNAL_SIZE * 2 * BATCH];
int16_t h_convolved_signal16[SIGNAL_SIZE * 2 * BATCH];

float elapsed_time, time_memcpy, time_kernel;   // timing variables

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
    LARGE_INTEGER li;
    if (!QueryPerformanceFrequency(&li))
        printf("QueryPerformanceFrequency failed!\n");

    PCFreq = (double)(li.QuadPart) / 1000.0;

    QueryPerformanceCounter(&li);
    CounterStart = li.QuadPart;
}
double GetCounter()
{
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    return (double)(li.QuadPart - CounterStart) / PCFreq;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { runTest(argc, argv); }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) 
{
    printf("[simpleCUFFT] is starting...\n");

  findCudaDevice(argc, (const char **)argv);

  // allocate and initialize an array of stream handles
  int nstreams = 4;
  hipStream_t* streams = (hipStream_t*)malloc(nstreams * sizeof(hipStream_t));
  for (int i = 0; i < nstreams; i++)
  {
      checkCudaErrors(hipStreamCreate(&(streams[i])));
  }

  // Allocate host memory for the signal
//  Complex *h_signal =
//      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));

  int mem_size = sizeof(hipfftComplex) * SIGNAL_SIZE * BATCH;

    // host arrays
  int16_t *h_PinnedSignal1;
  Complex *h_PinnedConvolvedSignal1;
  int16_t *h_PinnedSignal2;
  Complex *h_PinnedConvolvedSignal2;
  // allocate and initialize
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedSignal1, mem_size/2)); // host pinned 
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedConvolvedSignal1, mem_size)); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedSignal2, mem_size/2)); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedConvolvedSignal2, mem_size)); // host pinned
/*
  int16_t *h_signal1_i16, h_signal2_i16;
  checkCudaErrors(hipHostMalloc((void**)&h_signal1_i16, mem_size/2)); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&h_signal1_i16, mem_size / 2)); // host pinned
*/

  // Initialize the memory for the signal
  for (unsigned int i = 0; i < SIGNAL_SIZE * 2 * BATCH; i+=2) {
      h_PinnedSignal1[i] = 0;
      h_PinnedSignal1[i+1] = 0;
      h_PinnedSignal2[i] = 0;
      h_PinnedSignal2[i + 1] = 0;
  }
  
  for (unsigned int i = 0; i < 2 * BATCH; i += 2) {
      h_PinnedSignal1[i] = i+1;
      h_PinnedSignal1[i + 1] = 0;
      h_PinnedSignal2[i] = i + BATCH + 1; 
      h_PinnedSignal2[i + 1] = 0;
  }

  // Allocate device memory for signal
  int16_t* d_signal1_i16, *d_signal2_i16;
  Complex *d_signal1, *d_signal2;
  Complex *r_signal1, *r_signal2;
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal1_i16), mem_size/2));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal2_i16), mem_size / 2));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal1), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&r_signal1), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal2), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&r_signal2), mem_size));

  StartCounter();
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_signal1_i16, h_PinnedSignal1, mem_size / 2, hipMemcpyHostToDevice));
  //checkCudaErrors(hipMemcpy(d_signal2_i16, h_PinnedSignal2, mem_size / 2, hipMemcpyHostToDevice));
  double copytime = GetCounter();
  printf("---- Pinned Memory Copy time %ld Bytes is: %0.3f milliseconds \n", mem_size/2, copytime);
  printf("--------------------------------------------- \n");

  // create CUDA event handles
// use blocking sync
  hipEvent_t start_event, stop_event;
  int eventflags = hipEventBlockingSync;

  checkCudaErrors(hipEventCreateWithFlags(&start_event, eventflags));
  checkCudaErrors(hipEventCreateWithFlags(&stop_event, eventflags));

  StartCounter();
  checkCudaErrors(hipEventRecord(start_event, 0)); 
  // Copy host memory to device
  checkCudaErrors(hipMemcpyAsync(d_signal1_i16, h_PinnedSignal1, mem_size/2, hipMemcpyHostToDevice, streams[0]));
  checkCudaErrors(hipMemcpyAsync(d_signal2_i16, h_PinnedSignal2, mem_size/2, hipMemcpyHostToDevice, streams[1]));
  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));   // block until the event is actually recorded
  checkCudaErrors(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
  copytime = GetCounter();
  printf("memcopy:\t%.2f\n", time_memcpy);

  printf("---- Pinned Memory Async Copy time %ld Bytes is: %0.3f milliseconds \n", mem_size, copytime);
  printf("--------------------------------------------- \n");

  // Launch the Int16toFloat32 CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (SIGNAL_SIZE + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  
  StartCounter();
  Int16toFloat32 << <blocksPerGrid, threadsPerBlock >> > (d_signal1_i16, d_signal1, BATCH, SIGNAL_SIZE);
  copytime = GetCounter();
  printf("---- Int16toFloat32 kernel time is: %0.3f milliseconds \n", copytime);
  printf("--------------------------------------------- \n");
  Int16toFloat32 << <blocksPerGrid, threadsPerBlock >> > (d_signal2_i16, d_signal2, BATCH, SIGNAL_SIZE);

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  err = hipGetLastError();
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to launch Int16toFloat32 kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMemcpy(h_PinnedConvolvedSignal1, d_signal1, mem_size,
      hipMemcpyDeviceToHost));

  // CUFFT plan simple API
  hipfftHandle plan;
  //checkCudaErrors(hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 1));
  checkCudaErrors(hipfftCreate(&plan));
  int rank = 1;
  long long int n = SIGNAL_SIZE;
  long long int inembed[] = { 0 };
  long long int istride = 1;// BATCH;
  long long int idist = n;// 1;
  hipDataType inputtype = HIP_C_32F;
  long long int onembed[] = { 0 };
  long long int ostride = 1;
  long long int odist = n;
  hipDataType outputtype = HIP_C_32F;
  long long int batch = BATCH;
  size_t workSize;
  hipDataType executiontype = HIP_C_32F;

  checkCudaErrors(hipfftXtMakePlanMany(plan, rank, &n,
      inembed, istride, idist, inputtype,
      onembed, ostride, odist, outputtype,
      batch, &workSize,
      executiontype));

  // Transform signal and kernel
  printf("Transforming signal hipfftXtExec\n");
  
  // timer init
  hipEvent_t start, stop;
  float gpuTime = 0.0f;
  hipEventCreate(&start, 0);
  hipEventCreate(&stop, 0);
  hipEventRecord(start, 0);
  hipEventSynchronize(start);

  StartCounter();
  checkCudaErrors(hipMemcpyAsync(d_signal2_i16, h_PinnedSignal2, mem_size / 2, hipMemcpyHostToDevice, streams[1]));
  checkCudaErrors(hipfftXtExec(plan, d_signal1, r_signal1, HIPFFT_FORWARD));
  checkCudaErrors(hipMemcpyAsync(d_signal1_i16, h_PinnedSignal1, mem_size / 2, hipMemcpyHostToDevice, streams[0]));
  checkCudaErrors(hipfftXtExec(plan, d_signal2, r_signal2, HIPFFT_FORWARD));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  double ttime = GetCounter();

  hipEventElapsedTime(&gpuTime, start, stop);
  printf("---- time: %.10f milliseconds\n", gpuTime);
  printf("---- Execution time is: %0.3f milliseconds \n", ttime);
  printf("--------------------------------------------- \n");

  // Copy device memory to host
  checkCudaErrors(hipMemcpy(h_PinnedConvolvedSignal1, r_signal1, mem_size,
        hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_PinnedConvolvedSignal2, r_signal2, mem_size,
      hipMemcpyDeviceToHost));

  for (int i = 0; i < BATCH; i++) {
      printf("1) BATCH %d: %f : %f\n", i, h_PinnedConvolvedSignal1[n * i].x, h_PinnedConvolvedSignal1[n * i].y);
      printf("2) BATCH %d: %f : %f\n", i, h_PinnedConvolvedSignal2[n * i].x, h_PinnedConvolvedSignal2[n * i].y);
  }
  
  checkCudaErrors(hipMemcpy(h_PinnedConvolvedSignal1, d_signal1, mem_size,
      hipMemcpyDeviceToHost));

  // Check if kernel execution generated and error
   getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

  // Transform signal back
  printf("Transforming signal back hipfftExecC2C\n");
  checkCudaErrors(hipfftXtExec(plan, r_signal1, d_signal1, HIPFFT_BACKWARD));

  // Copy device memory to host
  checkCudaErrors(hipMemcpy(h_PinnedConvolvedSignal1, d_signal1, mem_size,
                             hipMemcpyDeviceToHost));

  
  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));

  // cleanup memory
  checkCudaErrors(hipFree(d_signal1));
  checkCudaErrors(hipFree(r_signal1));
  checkCudaErrors(hipFree(d_signal2));
  checkCudaErrors(hipFree(r_signal2));

  exit(EXIT_SUCCESS);
}


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////
/*
// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b,
                                                   int size, float scale) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = threadID; i < size; i += numThreads) {
    a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
  }
}
*/

// Int16 to Float32
__global__ void Int16toFloat32(int16_t *a, Complex *b, int batch, long long size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j=0;j<batch;j++) {
        for (int i = threadID; i < size; i += numThreads) {
            b[i + size * j].x = a[i * 2 * batch + j * 2];
            b[i + size * j].y = a[i * 2 * batch + 1 + j * 2];
        }
    }
}

