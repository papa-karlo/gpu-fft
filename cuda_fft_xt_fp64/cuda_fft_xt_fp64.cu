#include "hip/hip_runtime.h"
/** 
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

#include <xmmintrin.h>

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// includes, CUDA
#include <builtin_types.h>

// Complex data type
typedef float2  Complex;
typedef double2 DoubleComplex;

// Int16 to Float32
__global__ void Int16toFloat32(int16_t* a, Complex* b, int batch, long long size);

// Int16 to Float64
__global__ void Int16toFloat64(int16_t* a, DoubleComplex* b, int batch, long long size);


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

// The filter size is assumed to be a number smaller than the signal size
//#define SIGNAL_SIZE 50
//#define SIGNAL_SIZE 1024
//#define SIGNAL_SIZE 8192
//#define SIGNAL_SIZE 256
#define SIGNAL_SIZE 0x400000
#define BATCH       16

DoubleComplex h_signal[SIGNAL_SIZE * BATCH];
//DoubleComplex h_convolved_signal[SIGNAL_SIZE * BATCH];

//int16_t h_signal16[SIGNAL_SIZE * 2 * BATCH];
int16_t h_convolved_signal16[SIGNAL_SIZE * 2 * BATCH];


double copytime;
double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
    LARGE_INTEGER li;
    if (!QueryPerformanceFrequency(&li))
        printf("QueryPerformanceFrequency failed!\n");

    PCFreq = (double)(li.QuadPart) / 1000.0;

    QueryPerformanceCounter(&li);
    CounterStart = li.QuadPart;
}
double GetCounter()
{
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    return (double)(li.QuadPart - CounterStart) / PCFreq;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { runTest(argc, argv); }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  printf("[simpleCUFFT] is starting...\n");

  findCudaDevice(argc, (const char **)argv);

    // Allocate host memory for the signal
//  Complex *h_signal =
//      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));

  int mem_size = sizeof(hipfftDoubleComplex) * SIGNAL_SIZE * BATCH;

    // host arrays
  DoubleComplex *h_PinnedSignal, *h_PinnedConvolvedSignal;
  // allocate and initialize
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedSignal, mem_size)); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&h_PinnedConvolvedSignal, mem_size)); // host pinned

  int16_t *h_signal16;
  checkCudaErrors(hipHostMalloc((void**)&h_signal16, mem_size/4)); // host pinned

  // Initialize the memory for the signal
  for (unsigned int i = 0; i < SIGNAL_SIZE * 2 * BATCH; i+=2) {
      h_signal16[i] = 0;
      h_signal16[i+1] = 0;
  }
  
  for (unsigned int i = 0; i < 2 * BATCH; i += 2) {
      h_signal16[i] = i+1;
      h_signal16[i + 1] = 0;
  }

    // Allocate device memory for signal
  int16_t* d_signal_i16;
  DoubleComplex *d_signal;
  DoubleComplex *r_signal;
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal_i16), mem_size/4));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&r_signal), mem_size));
  
  StartCounter();
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_signal_i16, h_signal16, mem_size/4, hipMemcpyHostToDevice));
  copytime = GetCounter();
  printf("---- Copy time %ld Bytes is: %0.3f milliseconds \n", mem_size/4, copytime);
  printf("--------------------------------------------- \n");

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (SIGNAL_SIZE + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  
  StartCounter();
  Int16toFloat64 << <blocksPerGrid, threadsPerBlock >> > (d_signal_i16, d_signal, BATCH, SIGNAL_SIZE);
  copytime = GetCounter();
  printf("---- Int16toFloat64 kernel time is: %0.3f milliseconds \n", copytime);
  printf("--------------------------------------------- \n");
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  err = hipGetLastError();
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to launch Int16toFloat32 kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMemcpy(h_signal, d_signal, mem_size,
      hipMemcpyDeviceToHost));

  // CUFFT plan simple API
  hipfftHandle plan;
  checkCudaErrors(hipfftCreate(&plan));
  int rank = 1;
  long long int n = SIGNAL_SIZE;
  long long int inembed[] = { 0 };
  long long int istride = 1;// BATCH;
  long long int idist = n;// 1;
  hipDataType inputtype = HIP_C_64F;
  long long int onembed[] = { 0 };
  long long int ostride = 1;
  long long int odist = n;
  hipDataType outputtype = HIP_C_64F;
  long long int batch = BATCH;
  size_t workSize;
  hipDataType executiontype = HIP_C_64F;

  checkCudaErrors(hipfftXtMakePlanMany(plan, rank, &n,
      inembed, istride, idist, inputtype,
      onembed, ostride, odist, outputtype,
      batch, &workSize,
      executiontype));

  // Transform signal and kernel
  printf("Transforming signal hipfftXtExec\n");
  
  // Launch the Vector Add CUDA Kernel
  threadsPerBlock = 1024;
  blocksPerGrid = (SIGNAL_SIZE + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

  cycle:
  // timer init
  hipEvent_t start, stop;
  float gpuTime = 0.0f;
  hipEventCreate(&start, 0);
  hipEventCreate(&stop, 0);
  hipEventRecord(start, 0);
  hipEventSynchronize(start);

  StartCounter();

  checkCudaErrors(hipfftXtExec(plan, d_signal, r_signal, HIPFFT_FORWARD));

  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  double ttime = GetCounter();

  hipEventElapsedTime(&gpuTime, start, stop);
  printf("---- time: %.10f milliseconds\n", gpuTime);
  printf("---- Execution time is: %0.3f milliseconds \n", ttime);
  printf("--------------------------------------------- \n");

  goto cycle;

  // Copy device memory to host
  checkCudaErrors(hipMemcpy(h_signal, r_signal, mem_size,
        hipMemcpyDeviceToHost));

  for (int i = 0; i < BATCH; i++) {
      printf("BATCH %d: %f : %f\n", i, h_signal[SIGNAL_SIZE * i].x, h_signal[SIGNAL_SIZE * i].y);
  }
  
  checkCudaErrors(hipMemcpy(h_signal, d_signal, mem_size,
      hipMemcpyDeviceToHost));

  // Check if kernel execution generated and error
   getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");
    
  
  
  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));

  // cleanup memory
  checkCudaErrors(hipFree(d_signal));
  checkCudaErrors(hipFree(r_signal));

  exit(EXIT_SUCCESS);
}


// Int16 to Float32
__global__ void Int16toFloat32(int16_t *a, Complex *b, int batch, long long size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j=0;j<batch;j++) {
        for (int i = threadID; i < size; i += numThreads) {
            b[i + size * j].x = a[i * 2 * batch + j * 2];
            b[i + size * j].y = a[i * 2 * batch + 1 + j * 2];
        }
    }
}

// Int16 to Float64
__global__ void Int16toFloat64(int16_t* a, DoubleComplex* b, int batch, long long size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int j = 0; j < batch; j++) {
        for (int i = threadID; i < size; i += numThreads) {
            b[i + size * j].x = a[i * 2 * batch + j * 2];
            b[i + size * j].y = a[i * 2 * batch + 1 + j * 2];
        }
    }
}

